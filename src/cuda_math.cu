#include "../include/common.h"
#include "../include/cuda_math.h"

void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, double* alpha,
                   const double*& d_A, const double*& d_B, double* beta,
                   double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    hipblasDgemm(handle, transA, transB, M, N, K, alpha, d_A, M, d_B, K, beta,
                d_C, M);
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, double* alpha, const double*& d_A, const double*& d_B,
                   double* beta, double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    hipblasDgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1);
}
