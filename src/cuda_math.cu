#include "hip/hip_runtime.h"
#include "../include/common.h"
#include "../include/cuda_math.h"

void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, double* alpha,
                   const double*& d_A, int LDA, const double*& d_B, int LDB,
                   double* beta, double*& d_C, int LDC) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    CHECK_CUBLAS(hipblasDgemm(handle, transA, transB, M, N, K, alpha, d_A, LDA,
                             d_B, LDB, beta, d_C, LDC));
    MY_CHECK(hipDeviceSynchronize());
}
void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, float* alpha,
                   const float*& d_A, int LDA, const float*& d_B, int LDB,
                   float* beta, float*& d_C, int LDC) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    CHECK_CUBLAS(hipblasSgemm(handle, transA, transB, M, N, K, alpha, d_A, LDA,
                             d_B, LDB, beta, d_C, LDC));
    MY_CHECK(hipDeviceSynchronize());
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, double* alpha, const double*& d_A, const double*& d_B,
                   double* beta, double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    CHECK_CUBLAS(
        hipblasDgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1));
    MY_CHECK(hipDeviceSynchronize());

    // WHAT ABOUT SYNRONIZING THE DEVICE?
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, float* alpha, const float*& d_A, const float*& d_B,
                   float* beta, float*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    CHECK_CUBLAS(
        hipblasSgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1));
    // hipDeviceSynchronize();
    MY_CHECK(hipDeviceSynchronize());

    // WHAT ABOUT SYNRONIZING THE DEVICE?
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, double* matrix,
                                          const double* vector, double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        matrix[idx] += alpha * vector[idx % rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, float* matrix,
                                          const float* vector, float alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        matrix[idx] += alpha * vector[idx % rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, const double* in,
                                          const double* vector, double* out,
                                          double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = in[idx] + alpha * vector[idx / rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, const float* in,
                                          const float* vector, float* out,
                                          float alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = in[idx] + alpha * vector[idx / rows];
    }
}
__global__ void cuda_exponential(int rows, int cols, double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] = exp(in[idx]);
    }
}

__global__ void cuda_exponential(int rows, int cols, float* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] = exp(in[idx]);
    }
}

__global__ void cuda_divide_colwise(int rows, int cols, float* in,
                                    const float* vec) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] /= vec[idx / rows];
    }
}

__global__ void cuda_divide_colwise(int rows, int cols, double* in,
                                    const double* vec) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] /= vec[idx / rows];
    }
}

__global__ void cuda_relu(int rows, int cols, double* out, const double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = (in[idx] > 0) ? in[idx] : 0.;
    }
}

__global__ void cuda_relu(int rows, int cols, float* out, const float* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = (in[idx] > 0) ? in[idx] : 0.;
    }
}

__global__ void cuda_relu_backwards(int rows, int cols, const double* values,
                                    const double* grad_in, double* grad_out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        grad_out[idx] = (values[idx] > 0) ? grad_in[idx] : 0.;
    }
}

__global__ void cuda_relu_backwards(int rows, int cols, const float* values,
                                    const float* grad_in, float* grad_out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        grad_out[idx] = (values[idx] > 0) ? grad_in[idx] : 0.;
    }
}

__global__ void cuda_all_cross_entropy_losses(int rows, int cols,
                                              const double* prediction,
                                              const double* actual,
                                              double* losses) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        if (actual[linear] == 1) {
            losses[linear / rows] = -1 * log(prediction[linear]);
        }
    }
}

__global__ void cuda_all_cross_entropy_losses(int rows, int cols,
                                              const float* prediction,
                                              const float* actual,
                                              float* losses) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        if (actual[linear] == 1) {
            losses[linear / rows] = -1 * log(prediction[linear]);
        }
    }
}

__global__ void cuda_cross_entropy_gradient(int rows, int cols,
                                            const float* prediction,
                                            const float* actual,
                                            float* gradient) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        gradient[linear] = prediction[linear] - actual[linear];
    }
}

__global__ void cuda_cross_entropy_gradient(int rows, int cols,
                                            const double* prediction,
                                            const double* actual,
                                            double* gradient) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        gradient[linear] = prediction[linear] - actual[linear];
    }
}

// I NEED TO THINK HOW I DO SUCH A SUM BETTER - Reduction!!!
__global__ void cuda_sum_cross_entropy_losses(int obs, float* loss,
                                              const float* all_losses) {
    unsigned int linear = blockIdx.x * blockDim.x + threadIdx.x;
    if (linear == 0) {
        for (int i = 0; i < obs; ++i) loss[0] += all_losses[i];
    }
}

__global__ void cuda_sum_cross_entropy_losses(int obs, double* loss,
                                              const double* all_losses) {
    unsigned int linear = blockIdx.x * blockDim.x + threadIdx.x;
    if (linear == 0) {
        for (int i = 0; i < obs; ++i) loss[0] += all_losses[i];
    }
}

__global__ void cuda_matrix_addition_inplace(int rows, int cols,
                                             const float* d_A, float* d_B,
                                             const float alpha) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        // printf("row %d, col %d, linear %d, old_val %.3f, gradient %.3f\n",
        // row, col, linear, d_B[linear], d_A[linear]);
        d_B[linear] += alpha * d_A[linear];
    }
}

__global__ void cuda_matrix_addition_inplace(int rows, int cols,
                                             const double* d_A, double* d_B,
                                             const double alpha) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        d_B[linear] += alpha * d_A[linear];
    }
}

void add_vec_to_mat_colwise(int rows, int cols, double* matrix,
                            const double* vector, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, matrix, vector,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
    //cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, float* matrix,
                            const float* vector, float alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, matrix, vector,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
    //cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, const double* in,
                            const double* vector, double* out, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, in, vector, out,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
    //cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, const float* in,
                            const float* vector, float* out, float alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, in, vector, out,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
    //cudaDeviceSynronize();
}

void exponential(int rows, int cols, double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_exponential<<<grid, block>>>(rows, cols, in);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void exponential(int rows, int cols, float* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_exponential<<<grid, block>>>(rows, cols, in);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void divide_colwise(int rows, int cols, double* in, const double* vec) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_divide_colwise<<<grid, block>>>(rows, cols, in, vec);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void divide_colwise(int rows, int cols, float* in, const float* vec) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_divide_colwise<<<grid, block>>>(rows, cols, in, vec);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void relu(int rows, int cols, double* out, const double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu<<<grid, block>>>(rows, cols, out, in);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}
void relu(int rows, int cols, float* out, const float* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu<<<grid, block>>>(rows, cols, out, in);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void relu_backwards(int rows, int cols, const double* values,
                    const double* grad_in, double* grad_out) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu_backwards<<<grid, block>>>(rows, cols, values, grad_in, grad_out);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}
void relu_backwards(int rows, int cols, const float* values,
                    const float* grad_in, float* grad_out) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu_backwards<<<grid, block>>>(rows, cols, values, grad_in, grad_out);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void all_cross_entropy_losses(int rows, int cols, const double* prediction,
                              const double* actual, double* losses) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_all_cross_entropy_losses<<<grid, block>>>(rows, cols, prediction,
                                                   actual, losses);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void all_cross_entropy_losses(int rows, int cols, const float* prediction,
                              const float* actual, float* losses) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_all_cross_entropy_losses<<<grid, block>>>(rows, cols, prediction,
                                                   actual, losses);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void sum_cross_entropy_losses(int obs, float* loss, const float* all_losses) {
    dim3 block(256);
    dim3 grid((obs + block.x - 1) / block.x);
    cuda_sum_cross_entropy_losses<<<grid, block>>>(obs, loss, all_losses);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void sum_cross_entropy_losses(int obs, double* loss, const double* all_losses) {
    dim3 block(256);
    dim3 grid((obs + block.x - 1) / block.x);
    cuda_sum_cross_entropy_losses<<<grid, block>>>(obs, loss, all_losses);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}
void cross_entropy_gradient(int rows, int cols, const double* prediction,
                            const double* target, double* gradient) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_cross_entropy_gradient<<<grid, block>>>(rows, cols, prediction, target,
                                                 gradient);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}
void cross_entropy_gradient(int rows, int cols, const float* prediction,
                            const float* target, float* gradient) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_cross_entropy_gradient<<<grid, block>>>(rows, cols, prediction, target,
                                                 gradient);
    // hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void matrix_addition_inplace(int rows, int cols, const float* gradient,
                             float* parameters, const float alpha) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_matrix_addition_inplace<<<grid, block>>>(rows, cols, gradient,
                                                  parameters, alpha);
    // hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void matrix_addition_inplace(int rows, int cols, const double* gradient,
                             double* parameters, const double alpha) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_matrix_addition_inplace<<<grid, block>>>(rows, cols, gradient,
                                                  parameters, alpha);
    //hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}
