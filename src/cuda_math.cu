#include "hip/hip_runtime.h"
#include "../include/common.h"
#include "../include/cuda_math.h"

void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, double* alpha,
                   const double*& d_A, int LDA, const double*& d_B, int LDB,
                   double* beta, double*& d_C, int LDC) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    hipblasDgemm(handle, transA, transB, M, N, K, alpha, d_A, LDA, d_B, LDB,
                beta, d_C, LDC);
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, double* alpha, const double*& d_A, const double*& d_B,
                   double* beta, double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    hipblasDgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1);
    // WHAT ABOUT SYNRONIZING THE DEVICE?
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, double* matrix,
                                          const double* vector, double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        matrix[idx] += alpha * vector[idx % rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, const double* in,
                                          const double* vector, double* out,
                                          double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = in[idx] + alpha * vector[idx % rows];
    }
}

__global__ void cuda_exponential(int rows, int cols, double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] = exp(in[idx]);
    }
}

__global__ void cuda_divide_colwise(int rows, int cols, double* in,
                                    const double* vec) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] /= vec[idx / rows];
    }
}

void add_vec_to_mat_colwise(int rows, int cols, double* matrix,
                            const double* vector, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, matrix, vector,
                                               alpha);
    // cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, const double* in,
                            const double* vector, double* out, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, in, vector, out,
                                               alpha);
    // cudaDeviceSynronize();
}

void exponential(int rows, int cols, double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_exponential<<<grid, block>>>(rows, cols, in);
}

void divide_colwise(int rows, int cols, double* in, const double* vec) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_divide_colwise<<<grid, block>>>(rows, cols, in, vec);
}
