#include "hip/hip_runtime.h"
#include <hiprand.h>
#include <sys/time.h>
#include "../include/common.h"
#include "../include/cuda_math.h"
#include <float.h>
double cpuSecond() {
    struct timeval tp;
    gettimeofday(&tp, NULL);
    return ((double)tp.tv_sec + (double)tp.tv_usec * 1e-6);
}

void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, double* alpha,
                   const double*& d_A, int LDA, const double*& d_B, int LDB,
                   double* beta, double*& d_C, int LDC) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    CHECK_CUBLAS(hipblasDgemm(handle, transA, transB, M, N, K, alpha, d_A, LDA,
                             d_B, LDB, beta, d_C, LDC));
    // MY_CHECK(hipDeviceSynchronize());
}
void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, float* alpha,
                   const float*& d_A, int LDA, const float*& d_B, int LDB,
                   float* beta, float*& d_C, int LDC) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    CHECK_CUBLAS(hipblasSgemm(handle, transA, transB, M, N, K, alpha, d_A, LDA,
                             d_B, LDB, beta, d_C, LDC));
    // MY_CHECK(hipDeviceSynchronize());
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, double* alpha, const double*& d_A, const double*& d_B,
                   double* beta, double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    CHECK_CUBLAS(
        hipblasDgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1));
    // MY_CHECK(hipDeviceSynchronize());

    // WHAT ABOUT SYNRONIZING THE DEVICE?
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, float* alpha, const float*& d_A, const float*& d_B,
                   float* beta, float*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    CHECK_CUBLAS(
        hipblasSgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1));
    // hipDeviceSynchronize();
    // MY_CHECK(hipDeviceSynchronize());

    // WHAT ABOUT SYNRONIZING THE DEVICE?
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, double* matrix,
                                          const double* vector, double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        matrix[idx] += alpha * vector[idx % rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, float* matrix,
                                          const float* vector, float alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        matrix[idx] += alpha * vector[idx % rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, const double* in,
                                          const double* vector, double* out,
                                          double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = in[idx] + alpha * vector[idx / rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, const float* in,
                                          const float* vector, float* out,
                                          float alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = in[idx] + alpha * vector[idx / rows];
    }
}
__global__ void cuda_exponential(int rows, int cols, double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] = exp(in[idx]);
    }
}

__global__ void cuda_exponential(int rows, int cols, float* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] = exp(in[idx]);
    }
}

__global__ void cuda_divide_colwise(int rows, int cols, float* in,
                                    const float* vec) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] /= vec[idx / rows];
    }
}

__global__ void cuda_divide_colwise(int rows, int cols, double* in,
                                    const double* vec) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] /= vec[idx / rows];
    }
}

__global__ void cuda_relu(int rows, int cols, double* out, const double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = (in[idx] > 0) ? in[idx] : 0.;
    }
}

__global__ void cuda_relu(int rows, int cols, float* out, const float* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = (in[idx] > 0) ? in[idx] : 0.;
    }
}

__global__ void cuda_relu_backwards(int rows, int cols, const double* values,
                                    const double* grad_in, double* grad_out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        grad_out[idx] = (values[idx] > 0) ? grad_in[idx] : 0.;
    }
}

__global__ void cuda_relu_backwards(int rows, int cols, const float* values,
                                    const float* grad_in, float* grad_out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        grad_out[idx] = (values[idx] > 0) ? grad_in[idx] : 0.;
    }
}

__global__ void cuda_all_cross_entropy_losses(int rows, int cols,
                                              const double* prediction,
                                              const double* actual,
                                              double* losses) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        if (actual[linear] == 1) {
            losses[linear / rows] = -1 * log(prediction[linear]);
        }
    }
}

__global__ void cuda_all_cross_entropy_losses(int rows, int cols,
                                              const float* prediction,
                                              const float* actual,
                                              float* losses) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        if (actual[linear] == 1) {
            losses[linear / rows] = -1 * log(prediction[linear]);
        }
    }
}

__global__ void cuda_cross_entropy_gradient(int rows, int cols,
                                            const float* prediction,
                                            const float* actual,
                                            float* gradient) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        gradient[linear] = prediction[linear] - actual[linear];
    }
}

__global__ void cuda_cross_entropy_gradient(int rows, int cols,
                                            const double* prediction,
                                            const double* actual,
                                            double* gradient) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        gradient[linear] = prediction[linear] - actual[linear];
    }
}

// I NEED TO THINK HOW I DO SUCH A SUM BETTER - Reduction!!!
__global__ void cuda_sum_cross_entropy_losses(int obs, float* loss,
                                              const float* all_losses) {
    unsigned int linear = blockIdx.x * blockDim.x + threadIdx.x;
    if (linear == 0) {
        for (int i = 0; i < obs; ++i) loss[0] += all_losses[i];
    }
}

__global__ void cuda_sum_cross_entropy_losses(int obs, double* loss,
                                              const double* all_losses) {
    unsigned int linear = blockIdx.x * blockDim.x + threadIdx.x;
    if (linear == 0) {
        for (int i = 0; i < obs; ++i) loss[0] += all_losses[i];
    }
}

__global__ void cuda_matrix_addition_inplace(int rows, int cols,
                                             const float* d_A, float* d_B,
                                             const float alpha) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        // printf("row %d, col %d, linear %d, old_val %.3f, gradient %.3f\n",
        // row, col, linear, d_B[linear], d_A[linear]);
        d_B[linear] += alpha * d_A[linear];
    }
}

__global__ void cuda_matrix_addition_inplace(int rows, int cols,
                                             const double* d_A, double* d_B,
                                             const double alpha) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        d_B[linear] += alpha * d_A[linear];
    }
}

__global__ void multiply_ele(int rows, int cols, const float* d_A,
                             const float* d_B, float* d_C) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        d_C[linear] = d_A[linear] * d_B[linear];
    }
}

__global__ void multiply_ele(int rows, int cols, const double* d_A,
                             const double* d_B, double* d_C) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        d_C[linear] = d_A[linear] * d_B[linear];
    }
}

__global__ void masking(int rows, int cols, const float prob, float* d_A) {
    unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        d_A[linear] = (d_A[linear] < prob) ? 1 / prob : 0.;
    }
}

__global__ void masking(int rows, int cols, const double prob, double* d_A) {
    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int linear = row + col * rows;
    if ((row < rows) && (col < cols)) {
        d_A[linear] = (d_A[linear] < prob) ? 1 / prob : 0.;
    }
}

__global__ void CudaPoolBackwards(const float* top_diff, const float* mask,
                                  int window, int stride, int rows, int cols,
                                  int channels, int batches,
                                  float* bottom_diff) {
    int row = (blockIdx.x * blockDim.x + threadIdx.x);
    int col = (blockIdx.y * blockDim.y + threadIdx.y);
    int c = (blockIdx.z * blockDim.z + threadIdx.z);
    int out_height = (rows - window) / stride + 1;
    int out_width = (cols - window) / stride + 1;
    if (row < rows && col < cols && c < channels) {
        const int phstart = (row < window) ? 0 : (row + window) / stride + 1;
        const int phend = min(row / stride + 1, out_height);
        const int pwstart = (cols < window) ? 0 : (col + window) / stride + 1;
        const int pwend = min(col / stride + 1, out_width);
        const int idx = c * rows * cols + row * cols + col;
        for (int n = 0; n < batches; ++n) {
            dtype gradient = 0;
            for (int ph = phstart; ph < phend; ++ph) {
                for (int pw = pwstart; pw < pwend; ++pw) {
                    int li = out_width * (c * out_height + ph) + pw;
                    if (mask[li] == row * cols + col) {
                        gradient += top_diff[li];
                        // bottom_diff[idx] += top_diff[li];
                    }
                }
            }
            bottom_diff[idx] = gradient;
            mask += out_width * out_height * channels;
            top_diff += out_width * out_height * channels;
            bottom_diff += channels * rows * cols;
            // bottom_diff[n * channels * rows * cols + c * rows * cols +
        }
        // row * cols + col] = gradient;
    }
}

__global__ void MaxPoolBackward(const int nthreads, const dtype* const top_diff,
                                const float* const mask, const int num,
                                const int channels, const int height,
                                const int width, const int pooled_height,
                                const int pooled_width, const int window,
                                const int stride, dtype* const bottom_diff) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < nthreads) {
        // CUDA_KERNEL_LOOP(index, nthreads) {
        // find out the local index
        // find out the local offset
        const int w = index % width;
        const int h = (index / width) % height;
        const int c = (index / width / height) % channels;
        const int n = index / width / height / channels;
        const int phstart = (h < window) ? 0 : (h + window) / stride + 1;
        const int phend = min((h) / stride + 1, pooled_height);
        const int pwstart = (w < window) ? 0 : (w + window) / stride + 1;
        const int pwend = min((w) / stride + 1, pooled_width);
        dtype gradient = 0;
        const int offset = (n * channels + c) * pooled_height * pooled_width;
        const dtype* const top_diff_slice = top_diff + offset;
        const float* const mask_slice = mask + offset;
        for (int ph = phstart; ph < phend; ++ph) {
            for (int pw = pwstart; pw < pwend; ++pw) {
                if (mask_slice[ph * pooled_width + pw] == h * width + w) {
                    gradient += top_diff_slice[ph * pooled_width + pw];
                }
            }
        }
        bottom_diff[index] = gradient;
    }
}

__global__ void MaxPoolForward(int nthreads, const dtype* bottom_data, int num,
                               int channels, int height, int width,
                               int out_height, int out_width, int window,
                               int stride, dtype* top_data,
                               dtype* mask) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < nthreads) {
        const int pw = index % out_width;
        const int ph = (index / out_width) % out_height;
        const int c = (index / out_width / out_height) % channels;
        const int n = index / out_width / out_height / channels;
        int hstart = ph * stride;
        int wstart = pw * stride;
        const int hend = min(hstart + window, height);
        const int wend = min(wstart + window, width);
        hstart = max(hstart, 0);
        wstart = max(wstart, 0);
        dtype maxval = -FLT_MAX;
        int maxidx = -1;
        const dtype* bottom_slice =
            bottom_data + (n * channels + c) * height * width;
        for (int h = hstart; h < hend; ++h) {
            for (int w = wstart; w < wend; ++w) {
                if (bottom_slice[h * width + w] > maxval) {
                    maxidx = h * width + w;
                    maxval = bottom_slice[maxidx];
                }
            }
        }
        top_data[index] = maxval;
        mask[index] = maxidx;
    }
}

__global__ void CudaPooling(float const* inp, int window, int stride, int rows,
                            int cols, int channels, int batches, float* out,
                            float* mask) {
    int ph = (blockIdx.x * blockDim.x + threadIdx.x);
    int pw = (blockIdx.y * blockDim.y + threadIdx.y);
    int c = (blockIdx.z * blockDim.z + threadIdx.z);
    int out_height = (rows - window) / stride + 1;
    int out_width = (cols - window) / stride + 1;
    if (ph < out_height && pw < out_width && c < channels) {
        for (int n = 0; n < batches; n++) {
            for (int i = 0; i < window; i++) {
                for (int j = 0; j < window; j++) {
                    int curRow = ph * stride + i;
                    int curCol = pw * stride + j;
                    int li = c * cols * rows + curRow * cols + curCol;
                    int lo = c * out_width * out_height + ph * out_width + pw;
                    if (inp[li] > out[lo]) {
                        out[lo] = inp[li];
                        mask[lo] = li % (cols * rows);
                    }
                }
            }
            inp += rows * cols * channels;
            out += out_width * out_height * channels;
            mask += out_width * out_height * channels;
        }
    }
}

__global__ void im2col_gpu_kernel(int numThreads, const dtype* data_im,
                                  const int height, const int width,
                                  const int kernel_h, const int kernel_w,
                                  const int pad, const int stride,
                                  const int out_height, const int out_width,
                                  dtype* data_col) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numThreads) {
        const int h_index = index / out_width;
        const int h_col = h_index % out_height;
        const int w_col = index % out_width;
        const int c_im = h_index / out_height;
        const int c_col = c_im * kernel_h * kernel_w;
        const int h_offset = h_col * stride - pad;
        const int w_offset = w_col * stride - pad;
        dtype* data_col_ptr = data_col;
        data_col_ptr += (c_col * out_height + h_col) * out_width + w_col;
        const dtype* data_im_ptr = data_im;
        data_im_ptr += (c_im * height + h_offset) * width + w_offset;
        for (int i = 0; i < kernel_h; ++i) {
            for (int j = 0; j < kernel_w; ++j) {
                int h_im = h_offset + i;
                int w_im = w_offset + j;
                *data_col_ptr =
                    (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width)
                        ? data_im_ptr[i * width + j]
                        : 0;
                data_col_ptr += out_height * out_width;
            }
        }
    }
}

__global__ void col2im_gpu_kernel(int numThreads, const dtype* data_col,
                                  int height, int width, int channels,
                                  int kernel_h, int kernel_w, const int pad,
                                  const int stride, const int height_col,
                                  const int width_col, dtype* data_im) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < numThreads) {
        dtype val = 0;
        int w_im = index % width + pad;
        int h_im = (index / width) % height + pad;
        int c_im = index / (width * height);
        // compute the start and end of the output
        int w_col_start =
            (w_im < kernel_w) ? 0 : (w_im - kernel_w) / stride + 1;
        int w_col_end = min(w_im / stride + 1, width_col);
        int h_col_start =
            (h_im < kernel_h) ? 0 : (h_im - kernel_h) / stride + 1;
        int h_col_end = min(h_im / stride + 1, height_col);
        for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
            for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
                int h_k = (h_im - h_col * stride);
                int w_k = (w_im - w_col * stride);
                // if (h_k % dilation_h == 0 && w_k % dilation_w == 0) {
                // h_k /= dilation_h;
                // w_k /= dilation_w;
                int data_col_index =
                    (((c_im * kernel_h + h_k) * kernel_w + w_k) * height_col +
                     h_col) *
                        width_col +
                    w_col;
                val += data_col[data_col_index];
            }
        }
        data_im[index] = val;
    }
}

void add_vec_to_mat_colwise(int rows, int cols, double* matrix,
                            const double* vector, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, matrix, vector,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
    // cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, float* matrix,
                            const float* vector, float alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, matrix, vector,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
    // cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, const double* in,
                            const double* vector, double* out, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, in, vector, out,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
    // cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, const float* in,
                            const float* vector, float* out, float alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, in, vector, out,
                                               alpha);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
    // cudaDeviceSynronize();
}

void exponential(int rows, int cols, double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_exponential<<<grid, block>>>(rows, cols, in);
    MY_CHECK(hipPeekAtLastError());
    MY_CHECK(hipDeviceSynchronize());
}

void exponential(int rows, int cols, float* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_exponential<<<grid, block>>>(rows, cols, in);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void divide_colwise(int rows, int cols, double* in, const double* vec) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_divide_colwise<<<grid, block>>>(rows, cols, in, vec);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void divide_colwise(int rows, int cols, float* in, const float* vec) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_divide_colwise<<<grid, block>>>(rows, cols, in, vec);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void relu(int rows, int cols, double* out, const double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu<<<grid, block>>>(rows, cols, out, in);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}
void relu(int rows, int cols, float* out, const float* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu<<<grid, block>>>(rows, cols, out, in);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void relu_backwards(int rows, int cols, const double* values,
                    const double* grad_in, double* grad_out) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu_backwards<<<grid, block>>>(rows, cols, values, grad_in, grad_out);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}
void relu_backwards(int rows, int cols, const float* values,
                    const float* grad_in, float* grad_out) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu_backwards<<<grid, block>>>(rows, cols, values, grad_in, grad_out);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void all_cross_entropy_losses(int rows, int cols, const double* prediction,
                              const double* actual, double* losses) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_all_cross_entropy_losses<<<grid, block>>>(rows, cols, prediction,
                                                   actual, losses);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void all_cross_entropy_losses(int rows, int cols, const float* prediction,
                              const float* actual, float* losses) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_all_cross_entropy_losses<<<grid, block>>>(rows, cols, prediction,
                                                   actual, losses);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void sum_cross_entropy_losses(int obs, float* loss, const float* all_losses) {
    dim3 block(256);
    dim3 grid((obs + block.x - 1) / block.x);
    cuda_sum_cross_entropy_losses<<<grid, block>>>(obs, loss, all_losses);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void sum_cross_entropy_losses(int obs, double* loss, const double* all_losses) {
    dim3 block(256);
    dim3 grid((obs + block.x - 1) / block.x);
    cuda_sum_cross_entropy_losses<<<grid, block>>>(obs, loss, all_losses);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}
void cross_entropy_gradient(int rows, int cols, const double* prediction,
                            const double* target, double* gradient) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_cross_entropy_gradient<<<grid, block>>>(rows, cols, prediction, target,
                                                 gradient);
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}
void cross_entropy_gradient(int rows, int cols, const float* prediction,
                            const float* target, float* gradient) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_cross_entropy_gradient<<<grid, block>>>(rows, cols, prediction, target,
                                                 gradient);
    // hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void matrix_addition_inplace(int rows, int cols, const float* gradient,
                             float* parameters, const float alpha) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_matrix_addition_inplace<<<grid, block>>>(rows, cols, gradient,
                                                  parameters, alpha);
    // hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void matrix_addition_inplace(int rows, int cols, const double* gradient,
                             double* parameters, const double alpha) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    cuda_matrix_addition_inplace<<<grid, block>>>(rows, cols, gradient,
                                                  parameters, alpha);
    // hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
    // MY_CHECK(hipDeviceSynchronize());
}

void multiply_elementwise(int rows, int cols, const float* d_A,
                          const float* d_B, float* d_C) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    multiply_ele<<<grid, block>>>(rows, cols, d_A, d_B, d_C);
    hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
}

void multiply_elementwise(int rows, int cols, const double* d_A,
                          const double* d_B, double* d_C) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    multiply_ele<<<grid, block>>>(rows, cols, d_A, d_B, d_C);
    hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
}

void cuda_masking(int rows, int cols, const float prob, float* d_A) {
    dim3 block(16, 16);
    dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y);
    masking<<<grid, block>>>(rows, cols, prob, d_A);
    hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
}

void cuda_masking(int rows, int cols, const double prob, double* d_A) {
    dim3 block(16, 16);
    dim3 grid((rows + block.y - 1) / block.y, (cols + block.x - 1) / block.x);
    masking<<<grid, block>>>(rows, cols, prob, d_A);
    hipDeviceSynchronize();
    MY_CHECK(hipPeekAtLastError());
}

//void pooling_gpu(const float* bottom_data, int window, int stride, int rows,
                 //int cols, int channels, int batches, float* top_data,
                 //float* mask) {
    //if (((rows - window) % stride) or ((cols - window) % stride)) {
        //throw std::invalid_argument("Doesnt match");
    //}
    //dim3 block(16, 16, 4);
    //dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y,
              //(channels + block.z - 1) / block.z);
    //CudaPooling<<<grid, block>>>(bottom_data, window, stride, rows, cols,
                                 //channels, batches, top_data, mask);
    //// MY_CHECK(hipDeviceSynchronize());
    //MY_CHECK(hipPeekAtLastError());
//}

void pooling_gpu(const float* bottom_data, int window, int stride, int rows,
                 int cols, int channels, int batches, float* top_data,
                 float* mask) {
    if (((rows - window) % stride) or ((cols - window) % stride)) {
        throw std::invalid_argument("Doesnt match");
    }
    int out_height = (rows - window) / stride + 1;
    int out_width = (cols - window) / stride + 1;
    dim3 block(512);
    int eles = out_height * out_width * channels * batches;
    dim3 grid((eles + block.x - 1) / block.x);
    MaxPoolForward<<<grid, block>>>(eles, bottom_data, batches, channels,
                                    rows, cols, out_height, out_width, window,
                                    stride, top_data, mask);
    // MY_CHECK(hipDeviceSynchronize());
//__global__ void MaxPoolForward(int nthreads, const dtype* bottom_data, int num,
                               //int channels, int height, int width,
                               //int out_height, int out_width, int window,
                               //int stride, int pad, dtype* top_data,
                               //dtype* mask) {
    MY_CHECK(hipPeekAtLastError());
}

// linear access is 20 % faster!
// void pooling_backward_gpu_my(const float* bottom_data, const float* mask,
// int window, int stride, int rows, int cols,
// int channels, int batches, float* dest) {
// if (((rows - window) % stride) or ((cols - window) % stride)) {
// throw std::invalid_argument("Doesnt match");
//}
// dim3 block(16, 16, 4);
// dim3 grid((rows + block.x - 1) / block.x, (cols + block.y - 1) / block.y,
//(channels + block.z - 1) / block.z);
// CudaPoolBackwards<<<grid, block>>>(bottom_data, mask, window, stride, rows,
// cols, channels, batches, dest);
// MY_CHECK(hipDeviceSynchronize());
// MY_CHECK(hipPeekAtLastError());
//}

void pooling_backward_gpu(const float* src, const float* mask, int window,
                          int stride, int rows, int cols, int channels,
                          int batches, float* dest) {
    if (((rows - window) % stride) or ((cols - window) % stride)) {
        throw std::invalid_argument("Doesnt match");
    }
    int out_height = (rows - window) / stride + 1;
    int out_width = (cols - window) / stride + 1;
    dim3 block(512);
    int eles = rows * cols * channels * batches;
    dim3 grid((eles + block.x - 1) / block.x);
    MaxPoolBackward<<<grid, block>>>(eles, src, mask, batches, channels, rows,
                                     cols, out_height, out_width, window,
                                     stride, dest);
    MY_CHECK(hipDeviceSynchronize());
    MY_CHECK(hipPeekAtLastError());
}

void im2col_gpu(const float* data_im, int channels, int height, const int width,
                int kernel_h, const int kernel_w, int pad, int stride,
                float* data_col) {
    // We are going to launch channels * height_col * width_col kernels, each
    // kernel responsible for copying a single-channel grid.
    int out_height = (height + 2 * pad - kernel_h) / stride + 1;
    int out_width = (width + 2 * pad - kernel_w) / stride + 1;
    int numThreads = channels * out_height * out_width;
    dim3 block(512);
    dim3 grid((numThreads + block.x - 1) / block.x);
    im2col_gpu_kernel<<<grid, block>>>(numThreads, data_im, height, width,
                                       kernel_h, kernel_w, pad, stride,
                                       out_height, out_width, data_col);
    MY_CHECK(hipDeviceSynchronize());
    MY_CHECK(hipPeekAtLastError());
}

void col2im_gpu(const dtype* data_col, int channels, int height, int width,
                int kernel_h, int kernel_w, int pad, int stride,
                dtype* data_im) {
    int out_height = (height + 2 * pad - kernel_h) / stride + 1;
    int out_width = (width + 2 * pad - kernel_w) / stride + 1;
    int numThreads = channels * height * width;
    dim3 block(512);
    dim3 grid((numThreads + block.x - 1) / block.x);
    // To avoid involving atomic operations, we will launch one kernel per
    // bottom dimension, and then in the kernel add up the top dimensions.
    // NOLINT_NEXT_LINE(whitespace/operators)
    col2im_gpu_kernel<<<grid, block>>>(numThreads, data_col, height, width,
                                       channels, kernel_h, kernel_w, pad,
                                       stride, out_height, out_width, data_im);
    MY_CHECK(hipDeviceSynchronize());
    MY_CHECK(hipPeekAtLastError());
}
