#include "hip/hip_runtime.h"
#include "../include/common.h"
#include "../include/cuda_math.h"

void my_cuda_Dgemm(hipblasHandle_t handle, hipblasOperation_t transA,
                   hipblasOperation_t transB, int M, int N, int K, double* alpha,
                   const double*& d_A, int LDA, const double*& d_B, int LDB,
                   double* beta, double*& d_C, int LDC) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    hipblasDgemm(handle, transA, transB, M, N, K, alpha, d_A, LDA, d_B, LDB,
                beta, d_C, LDC);
}

void my_cuda_Dgemv(hipblasHandle_t handle, hipblasOperation_t transA, int M,
                   int N, double* alpha, const double*& d_A, const double*& d_B,
                   double* beta, double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    hipblasDgemv(handle, transA, M, N, alpha, d_A, M, d_B, 1, beta, d_C, 1);
    // WHAT ABOUT SYNRONIZING THE DEVICE?
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, double* matrix,
                                          const double* vector, double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        matrix[idx] += alpha * vector[idx % rows];
    }
}

__global__ void add_vec_to_mat_colwise_cu(int rows, int cols, const double* in,
                                          const double* vector, double* out,
                                          double alpha) {
    // get the current element index for the thread
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        out[idx] = in[idx] + alpha * vector[idx / rows];
    }
}

__global__ void cuda_exponential(int rows, int cols, double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] = exp(in[idx]);
    }
}

__global__ void cuda_divide_colwise(int rows, int cols, double* in,
                                    const double* vec) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        in[idx] /= vec[idx / rows];
    }
}

__global__ void cuda_relu(int rows, int cols, double* out, const double* in) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        if (in[idx] > 0)
            out[idx] = in[idx];
        else
            out[idx] = 0;
    }
}

__global__ void cuda_relu_backwards(int rows, int cols, const double* values,
       const double* grad_in, double* grad_out) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < rows * cols) {
        if (values[idx] > 0) {
            grad_out[idx] = grad_in[idx];
        } else {
           grad_out[idx] = 0;
        }
    }
}

void add_vec_to_mat_colwise(int rows, int cols, double* matrix,
                            const double* vector, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, matrix, vector,
                                               alpha);
    // cudaDeviceSynronize();
}

void add_vec_to_mat_colwise(int rows, int cols, const double* in,
                            const double* vector, double* out, double alpha) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    add_vec_to_mat_colwise_cu<<<grid, block>>>(rows, cols, in, vector, out,
                                               alpha);
    // cudaDeviceSynronize();
}

void exponential(int rows, int cols, double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_exponential<<<grid, block>>>(rows, cols, in);
}

void divide_colwise(int rows, int cols, double* in, const double* vec) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_divide_colwise<<<grid, block>>>(rows, cols, in, vec);
}

void relu(int rows, int cols, double* out, const double* in) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu<<<grid, block>>>(rows, cols, out, in);
}

void relu_backwards(int rows, int cols, const double* values, 
        const double* grad_in, double* grad_out) {
    dim3 block(256);
    dim3 grid((rows * cols + block.x - 1) / block.x);
    cuda_relu_backwards<<<grid, block>>>(rows, cols, values, grad_in, grad_out);
}
