#include "../include/common.h"
#include "../include/cuda_math.h"

void cuda_mathmult(hipblasHandle_t handle, int M, int N, int K, double* alpha,
                    const double*& d_A, const double*& d_B, double* beta,
                   double*& d_C) {
    // M defines the number of rows in Matrix A and C
    // N Defines the number of columns of the Matrix B and C
    // K defiens the number of columns of the Matrhx A and rows of Matix B
    hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, alpha, d_A, M, d_B,
                K, beta, d_C, M);
}
